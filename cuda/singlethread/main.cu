#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hip/hip_cooperative_groups.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <string.h>

#include "./files/bnetflix_psdd_gpu_cuda_4.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 */
__global__ void
main_ac(float *A, int nIter) { 
  ac(A, nIter); 
}

int 
main(int argc, char **argv)
{
    // nIter 
    int nIter = getCmdLineArgumentInt(argc, (const char **)argv, "nIter");
    
    size_t size_a= sizeof(float)* SIZE_OF_IN;

    // Allocate the device input vector A
    float *d_A = NULL;
    gpuErrchk(hipMalloc((void **)&d_A, size_a));


    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    gpuErrchk(hipMemcpy(d_A, h_A, size_a, hipMemcpyHostToDevice));

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid= BLOCKS_PER_GRID;
    struct timeval t1, t2;
    gettimeofday(&t1, 0);

    main_ac<<<blocksPerGrid, threadsPerBlock>>>(d_A, nIter);

    // FInish execution of kernel
    hipDeviceSynchronize();

    gettimeofday(&t2, 0);
    
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    
    printf("Time of kernel:  %3.4f ms \n", time);

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    gpuErrchk(hipGetLastError());

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    gpuErrchk(hipMemcpy(h_A, d_A, size_a, hipMemcpyDeviceToHost));
    
    for (int i=0; i< 4; i++) {
      printf("%d , %f | ", i, h_A[i]);
    }

    gpuErrchk(hipFree(d_A));

    printf("Done!\n");
    return 0;
}
